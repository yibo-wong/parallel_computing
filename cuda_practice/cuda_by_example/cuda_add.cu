
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 10

__global__ void add(int *a, int *b, int *c)
{
    int tid = blockIdx.x; // this thread handles the data at its thread id
    if (tid < N)
    {
        c[tid] = a[tid] + b[tid];
    }
    printf("%d", c[tid]);
}

int main(void)
{
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;

    // allocate the memory on the GPU
    hipMalloc((void **)&dev_a, N * sizeof(int));
    hipMalloc((void **)&dev_b, N * sizeof(int));
    hipMalloc((void **)&dev_c, N * sizeof(int));

    // fill the arrays 'a' and 'b' on the CPU
    for (int i = 0; i < N; i++)
    {
        a[i] = -i;
        b[i] = i * i;
    }

    // copy the arrays 'a' and 'b' to the GPU
    hipMemcpy(dev_a, a, N * sizeof(int),
               hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(int),
               hipMemcpyHostToDevice);

    add<<<N, 1>>>(dev_a, dev_b, dev_c);
    hipError_t error = hipGetLastError();
    printf("CUDA error: %s\n", hipGetErrorString(error));

    // copy the array 'c' back from the GPU to the CPU
    hipMemcpy(c, dev_c, N * sizeof(int),
               hipMemcpyDeviceToHost);

    // display the results
    for (int i = 0; i < N; i++)
    {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    // free the memory allocated on the GPU
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    return 0;
}
