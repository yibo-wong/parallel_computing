#include <iostream>
#include <math.h>
#include <hip/hip_runtime.h>
using namespace std;
int main() {
    int count = 0;
 	hipGetDeviceCount(&count);
	cout <<"当前计算机包含GPU数为"<< count << endl;
    hipError_t err = hipGetDeviceCount(&count);
    if (err != hipSuccess) 
	    printf("%s\n", hipGetErrorString(err));


    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("Device Number: %d\n", 0);
    cout << "当前设备名字为" << prop.name << endl;
	cout << "GPU全局内存总量为" << prop.totalGlobalMem << endl;
	cout << "单个线程块中包含的线程数最多为" << prop.maxThreadsPerBlock << endl;
  
}

